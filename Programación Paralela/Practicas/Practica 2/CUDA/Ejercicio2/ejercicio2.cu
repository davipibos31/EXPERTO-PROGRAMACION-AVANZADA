#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define TILE_SIZE 256

__global__ void saxpy(float alpha, float* x, float* y, float* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
        result[i] = alpha * x[i] + y[i];
}

int main()
{
    float alpha = 2.0f;
    float* h_x = (float*)malloc(N * sizeof(float));
    float* h_y = (float*)malloc(N * sizeof(float));
    float* h_result = (float*)malloc(N * sizeof(float));
    float* d_x;
    float* d_y;
    float* d_result;

    for (int i = 0; i < N; i++)
    {
        h_x[i] = i;
        h_y[i] = i + 1;
    }

    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));
    hipMalloc((void**)&d_result, N * sizeof(float));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE);

    saxpy<<<grid, block>>>(alpha, d_x, d_y, d_result);

    hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        printf("result[%d] = %.2f\n", i, h_result[i]);

    free(h_x);
    free(h_y);
    free(h_result);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    return 0;
}
