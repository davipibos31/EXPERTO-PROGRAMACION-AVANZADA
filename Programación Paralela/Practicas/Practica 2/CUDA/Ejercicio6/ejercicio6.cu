#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define EPSILON 0.0001

//@Rellenar: Kernel que implementa el método de Jacobi
__global__ void jacobi(float *d_A, float *d_B, float *d_C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > 0 && row < N - 1 && col > 0 && col < N - 1)
    {
        d_C[row * N + col] = 0.2f * (d_B[row * N + col] + d_A[(row - 1) * N + col] + d_A[row * N + col - 1] + d_A[(row + 1) * N + col] + d_A[row * N + col + 1]);
    }
}

//@Rellenar: Función para inicializar las matrices A y B
void initializeMatrices(float *h_A, float *h_B)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
            {
                h_A[i * N + j] = 0.0f;  // Valores fijos en los límites
                h_B[i * N + j] = 0.0f;
            }
            else
            {
                h_A[i * N + j] = 1.0f;  // Valor inicial arbitrario en el interior
                h_B[i * N + j] = 1.0f;
            }
        }
    }
}

//@Rellenar: Función para imprimir una matriz
void printMatrix(float *matrix)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.2f ", matrix[i * N + j]);
        }
        printf("\n");
    }
}

//@Rellenar: Función para verificar la convergencia del método
int checkConvergence(float *h_A, float *h_B)
{
    for (int i = 1; i < N - 1; i++)
    {
        for (int j = 1; j < N - 1; j++)
        {
            if (fabs(h_A[i * N + j] - h_B[i * N + j]) > EPSILON)
            {
                return 0;  // No ha convergido
            }
        }
    }
    return 1;  // Ha convergido
}

int main(int argc, char **argv)
{
    float *h_A = (float *)malloc(N * N * sizeof(float));
    float *h_B = (float *)malloc(N * N * sizeof(float));
    float *d_A;
    float *d_B;
    float *temp;

    //@Rellenar: Indica el tamaño de bloque en hilos.
    dim3 block(16, 16);
    //@Rellenar: Parámetros del kernel. Hilos por bloque, bloques por Grid.
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));

    initializeMatrices(h_A, h_B);
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    int iterations = 0;
    while (!checkConvergence(h_A, h_B))
    {
        jacobi<<<grid, block>>>(d_A, d_B, d_A);
        iterations++;
        temp = d_A;
        d_A = d_B;
        d_B = temp;
    }

    hipMemcpy(h_A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Converged after %d iterations.\n", iterations);
    printf("Final matrix:\n");
    printMatrix(h_A);

    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
