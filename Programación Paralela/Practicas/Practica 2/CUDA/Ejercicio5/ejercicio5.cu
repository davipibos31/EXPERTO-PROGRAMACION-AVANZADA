#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (1024)
#define TILE_SIZE 16

//@Rellenar: Kernel que desarrolla la suma de dos matrices
__global__ void sumaMatrices(float *d_a, float *d_b, float *d_c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    d_c[row * N + col] = d_a[row * N + col] + d_b[row * N + col];
}

int main(int argc, char **argv)
{
    float *h_a = (float *)malloc(N * N * sizeof(float));
    float *h_b = (float *)malloc(N * N * sizeof(float));
    float *h_c = (float *)malloc(N * N * sizeof(float));
    float *d_a;
    float *d_b;
    float *d_c;

    //@Rellenar: Indica el tamaño de bloque en hilos.
    dim3 block(TILE_SIZE, TILE_SIZE);
    //@Rellenar: Parámetros del kernel. Hilos por bloque, bloques por Grid.
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    hipMalloc((void **)&d_a, N * N * sizeof(float));
    hipMalloc((void **)&d_b, N * N * sizeof(float));
    hipMalloc((void **)&d_c, N * N * sizeof(float));

    // Inicializar matrices h_a y h_b
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_a[i * N + j] = i + j;
            h_b[i * N + j] = i - j;
        }
    }

    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    //@Rellenar: Parámetros del kernel. Hilos por bloque, bloques por Grid.
    sumaMatrices<<<grid, block>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Imprimir matriz resultante h_c
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.2f ", h_c[i * N + j]);
        }
        printf("\n");
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
