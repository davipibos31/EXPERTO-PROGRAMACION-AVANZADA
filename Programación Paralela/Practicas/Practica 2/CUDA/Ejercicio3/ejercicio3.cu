#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define TILE_SIZE 256

__global__ void reductionSum(int* input, int* output)
{
    __shared__ int sharedData[TILE_SIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    sharedData[tid] = (i < N) ? input[i] : 0;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
            sharedData[tid] += sharedData[tid + s];

        __syncthreads();
    }

    if (tid == 0)
        output[blockIdx.x] = sharedData[0];
}

int main()
{
    int* h_input = (int*)malloc(N * sizeof(int));
    int* h_output = (int*)malloc(N * sizeof(int));
    int* d_input;
    int* d_output;

    for (int i = 0; i < N; i++)
        h_input[i] = i + 1;

    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output, N * sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE);

    reductionSum<<<grid, block>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, grid.x * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < grid.x; i++)
        sum += h_output[i];

    printf("Sum: %d\n", sum);

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
