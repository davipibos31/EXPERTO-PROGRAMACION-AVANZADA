#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define TILE_SIZE 256

__global__ void stencil1D(float* y, float* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    __shared__ float tile[TILE_SIZE + 2];

    int ti = tx + 1;
    if (i >= N)
        tile[ti] = 0.0f;
    else
        tile[ti] = y[i];

    if (tx == 0)
        tile[ti - 1] = (i > 0) ? y[i - 1] : 0.0f;

    if (tx == blockDim.x - 1)
        tile[ti + 1] = (i < N - 1) ? y[i + 1] : 0.0f;

    __syncthreads();

    if (i < N)
        result[i] = (tile[ti - 1] * tile[ti] * tile[ti + 1]) / 3;
}

int main()
{
    float* h_y = (float*)malloc(N * sizeof(float));
    float* h_result = (float*)malloc(N * sizeof(float));
    float* d_y;
    float* d_result;

    for (int i = 0; i < N; i++)
        h_y[i] = i + 1;

    hipMalloc((void**)&d_y, N * sizeof(float));
    hipMalloc((void**)&d_result, N * sizeof(float));

    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE);

    stencil1D<<<grid, block>>>(d_y, d_result);

    hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        printf("y[%d] = %.2f\n", i, h_result[i]);

    free(h_y);
    free(h_result);
    hipFree(d_y);
    hipFree(d_result);

    return 0;
}
