
#include <hip/hip_runtime.h>
#include <omp.h>

int main() {
   double a[1000], b[1000], c[1000];

   #pragma omp parallel for
   for(int i = 0; i < 1000; i++) {
      a[i] = b[i] + c[i];
   }

   return 0;
}
