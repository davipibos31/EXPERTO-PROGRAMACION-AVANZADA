#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
  // Reserva e inicializa memoria en la CPU
  float *h_A = new float[N], *h_B = new float[N];

  // Reserva memoria en la GPU
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, N * sizeof(float));
  hipMalloc((void**)&d_B, N * sizeof(float));
  hipMalloc((void**)&d_C, N * sizeof(float));

  // Copia host memory en device
  hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

  // Ejecuta el kernel en ceil(N/256) blocks de 256 threads cada uno
  vecAdd<<<ceil(N/256.0), 256>>>(d_A, d_B, d_C, N);

  // Copia el resultado de vuelta a la CPU
  hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  // Libera memoria de la GPU
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Libera memoria de la CPU
  delete[] h_A;
  delete[] h_B;

  return 0;
}
