#include <assert.h>
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>


#define N 16384
#define TAMBLOCK 256
#define NUMITER 4096 //numero de iteraciones
__global__ void stencil1D(float *d_a, float *d_b){
	
	int indice = blockIdx.x * blockDim.x + threadIdx.x;

	if (indice > 0 && indice < (N-1))
	d_b[indice] = (d_a[indice] + d_a[indice - 1] + d_a[indice + 1])/3.0f;
	
}


int main()
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int memsize = N*sizeof(float);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

	h_a = (float *)malloc(memsize);

	if (h_a == NULL){
		fprintf(stderr,"Error al reservar memoria en el host\n");
		return EXIT_FAILURE;
	}

	h_b = (float *)malloc(memsize);

	if (h_b == NULL){
		fprintf(stderr, "Error al reservar memoria en el host\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMalloc(&d_a, memsize);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al reservar memoria en el device\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMalloc(&d_b, memsize);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al reservar memoria en el device\n");
		return EXIT_FAILURE;
	}

	//los bordes deberiamos de haberlos inicialilzado (h_b m�s que nada porque en el array el primer hueco no hay nada): 150 70 70 70 ... 70 150
	h_a[0] = h_a[N-1] = 150.0f; /*No inicializo h_b por que estar�a copiando todo el rato*/
	for (int i = 1; i < (N-1); ++i){
		h_a[i] = 70.0f;
	}

	cudaStatus = hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al transferir memoria al device\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMemcpy(d_b, h_a, memsize, hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al transferir memoria al device\n");
		return EXIT_FAILURE;
	}

	hipEvent_t start, stop; 

	hipEventCreateWithFlags(&start, NULL);
	hipEventCreateWithFlags(&stop, NULL);

	dim3 block(N/TAMBLOCK);
	dim3 hilos(TAMBLOCK);

	float *aux = NULL; 
	hipEventRecord(start, 0);
	for (int i = 0; i < NUMITER; ++i) {
		stencil1D<<<block, hilos>>>(d_a, d_b);
		//cudaThreadSynchronize();
		aux = d_a;
		d_a = d_b;
		d_b = aux; 
	}
	hipEventRecord (stop, 0);
	hipEventSynchronize(stop);
	float msec =0.0f;
	hipEventElapsedTime(&msec, start, stop);
	printf ("El tiempo de ejecucion es %f msec\n", msec);

	hipMemcpy (h_a, d_a, memsize, hipMemcpyDeviceToHost);
	/*for (int i = 0; i<N; ++i) {
			printf ("%f, ", h_a[i]);
		}
		printf ("\n");
	*/

	 
    hipFree(d_a);
    hipFree(d_b);
	free (h_a);
	free (h_b);
 
	system ("PAUSE");
    return cudaStatus;
}
