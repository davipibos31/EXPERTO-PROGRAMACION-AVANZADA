#include <assert.h>
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>


#define N 6
#define TAMBLOCK 2
#define NUMITER 100 //numero de iteraciones
__global__ void stencil1D(float *d_a, float *d_b){
	
	int indice = blockIdx.x * blockDim.x + threadIdx.x;

	if (indice > 0 && indice < (N-1))
	d_b[indice] = (d_a[indice] + d_a[indice - 1] + d_a[indice + 1])/3.0f;
	
}

__global__ void stencil1DconShared (float *d_a, float *d_b)
{

	__shared__ float s_a[TAMBLOCK];

	int indice = blockIdx.x * blockDim.x + threadIdx.x + 1;

	s_a[threadIdx.x]=d_a[indice];
	
	__syncthreads();

	float movLeft = (threadIdx.x==0)? d_a[indice-1]: s_a[threadIdx.x-1];
	float movRight = (threadIdx.x==(TAMBLOCK-1))? d_a[indice+1] : s_a[threadIdx.x+1];

	d_b[indice] = (movLeft + s_a[threadIdx.x] + movRight)/3.0f;
	
}


int main()
{
	float *h_a, *h_b;
	float *d_a, *d_b;
	int memsize = N*sizeof(float);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

	h_a = (float *)malloc(memsize);

	if (h_a == NULL){
		fprintf(stderr,"Error al reservar memoria en el host\n");
		return EXIT_FAILURE;
	}

	h_b = (float *)malloc(memsize);

	if (h_b == NULL){
		fprintf(stderr, "Error al reservar memoria en el host\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMalloc(&d_a, memsize);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al reservar memoria en el device\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMalloc(&d_b, memsize);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al reservar memoria en el device\n");
		return EXIT_FAILURE;
	}

	//los bordes deberiamos de haberlos inicialilzado (h_b m�s que nada porque en el array el primer hueco no hay nada): 150 70 70 70 ... 70 150
	h_a[0] = h_a[N-1] = 150.0f; /*No inicializo h_b por que estar�a copiando todo el rato*/
	for (int i = 1; i < (N-1); ++i){
		h_a[i] = 70.0f;
	}

	cudaStatus = hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al transferir memoria al device\n");
		return EXIT_FAILURE;
	}

	cudaStatus = hipMemcpy(d_b, h_a, memsize, hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess){
		fprintf(stderr, "Error al transferir memoria al device\n");
		return EXIT_FAILURE;
	}

	hipEvent_t start, stop; 

	hipEventCreateWithFlags(&start, NULL);
	hipEventCreateWithFlags(&stop, NULL);

	dim3 block((N-2)/TAMBLOCK);
	dim3 hilos(TAMBLOCK);

	float *aux = NULL; 
	hipEventRecord(start, 0);
	for (int i = 0; i < NUMITER; ++i) {
		stencil1DconShared<<<block, hilos>>>(d_a, d_b);
		//cudaThreadSynchronize();
		aux = d_a;
		d_a = d_b;
		d_b = aux; 
	}
	hipEventRecord (stop, 0);
	hipEventSynchronize(stop);
	float msec =0.0f;
	hipEventElapsedTime(&msec, start, stop);
	printf ("El tiempo de ejecucion es %f msec\n", msec);

	hipMemcpy (h_a, d_a, memsize, hipMemcpyDeviceToHost);
	for (int i = 0; i<N; ++i) {
			printf ("%.2f, ", h_a[i]);
		}
	printf ("\n");
	

	 
    hipFree(d_a);
    hipFree(d_b);
	free (h_a);
	free (h_b);
 
	system ("PAUSE");
    return cudaStatus;
}
